
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void globalMax(int *a, int N, int* gl_max)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
	int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) 
    {
      int val = a[index];
      __shared__ int subMax;
      
      if (threadIdx.x == 0)
          subMax = 0;
      __syncthreads();     

      atomicMax(&subMax, val);
      __syncthreads();
     
      if (threadIdx.x == 0)  
	    atomicMax(gl_max, subMax);
    }
}

#define THREADS_PER_BLOCK 512

int main(int argc, char*argv[])
{
    int N = atoi(argv[1]);
    //assert(N>0 && N<=1000000);
    int *a;
	int *d_a;
    int *d_max;
	int size = N * sizeof( int );

	/* allocate space for device copies of a, max */

	hipMalloc( (void **) &d_a, size );
	hipMalloc( (void **) &d_max, sizeof(int) );


	/* allocate space for host copies of a, cpu_max, and setup input values */

	a = (int *)malloc( size );
    int cpu_max = 0;

	for( int i = 0; i < N; i++ )
	{
		a[i] = i;
	}

	/* copy inputs to device */
	/* fix the parameters needed to copy data to the device */
	hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
	hipMemcpy( d_max, &cpu_max, sizeof(int), hipMemcpyHostToDevice );


	/* launch the kernel on the GPU */
	/* insert the launch parameters to launch the kernel properly using blocks and threads */ 
	globalMax<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, N, d_max);


	/* copy result back to host */
	/* fix the parameters needed to copy data back to the host */
	hipMemcpy( &cpu_max, d_max, sizeof(int), hipMemcpyDeviceToHost );


	printf( "global max = %d\n", cpu_max);

	/* clean up */

	free(a);
	hipFree( d_a );
	hipFree( d_max );
	
	return 0;
} /* end main */
