

#include <hip/hip_runtime.h>
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__global__ void globalMax(int *a, int N, int* gl_max)
{
    /* insert code to calculate the index properly using blockIdx.x, blockDim.x, threadIdx.x */
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int val = a[index];
        atomicMax(gl_max, val);
    }
}

#define THREADS_PER_BLOCK 512

int main(int argc, char*argv[])
{
    assert(argc == 2);
    int N = atoi(argv[1]);
    assert(N>0 && N<=10000000);
    int *a;
    int *d_a;
    int *d_max;
    int size = N * sizeof( int );

    time_t t;
    srand((unsigned) time(&t));

    /* allocate space for device copies of a, max */

    hipMalloc( (void **) &d_a, size );
    hipMalloc( (void **) &d_max, sizeof(int) );


    /* allocate space for host copies of a, cpu_max, and setup input values */

    a = (int *)malloc( size );
    int cpu_max = 0;

    for( int i = 0; i < N; i++ )
    {
        a[i] = rand() % 50;
    }

    /* copy inputs to device */
    /* fix the parameters needed to copy data to the device */
    hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
    hipMemcpy( d_max, &cpu_max, sizeof(int), hipMemcpyHostToDevice );


    /* launch the kernel on the GPU */
    /* insert the launch parameters to launch the kernel properly using blocks and threads */ 
    globalMax<<< (N + (THREADS_PER_BLOCK-1)) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( d_a, N, d_max);


    /* copy result back to host */
    /* fix the parameters needed to copy data back to the host */
    hipMemcpy( &cpu_max, d_max, sizeof(int), hipMemcpyDeviceToHost );


    printf( "global max = %d\n", cpu_max);

    /* clean up */

    free(a);
    hipFree( d_a );
    hipFree( d_max );
    
    return 0;
} /* end main */
